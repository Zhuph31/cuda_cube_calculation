
#include <hip/hip_runtime.h>
#include <cmath>
#include <iostream>
#include <sys/time.h>
#include <unistd.h>
#include <unordered_map>
#include <vector>

// #define NUM_BANKS 32
// #define LOG_NUM_BANKS 5
// #define CONFLICT_FREE_OFFSET(n) ((n) >> LOG_NUM_BANKS)
int block_size = 1024;
int n_stream = 30;

class TimeCost {
  double get_timestamp() const {
    struct timeval tv;
    gettimeofday(&tv, NULL);
    return (double)tv.tv_usec / 1000000 + tv.tv_sec;
  }

  double start_ts;

public:
  TimeCost() { start_ts = get_timestamp(); }
  double get_elapsed() const { return get_timestamp() - start_ts; }
};

struct ExecRecord {
  ExecRecord() {}
  void print() const {}
};

struct ExecRecords {
  double cpu_record;
  struct GPURecords {
  } gpu_records;
};

inline void check_kernel_err() {
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf(stderr, "Error: kernel invoke failed, %s\n",
            hipGetErrorString(err));
    exit(-1);
  }
}

void print_cube(float ***cube, int n) {
  for (int i = 0; i < n; i++) {
    for (int j = 0; j < n; j++) {
      for (int k = 0; k < n; k++) {
        printf("%lf, ", cube[i][j][k]);
      }
      printf("\n");
    }
    printf("\n");
  }
}

void cpu_malloc_cube(float ****cube_ref, int n) {
  (*cube_ref) = (float ***)malloc(n * sizeof(float **));
  for (int i = 0; i < n; i++) {
    (*cube_ref)[i] = (float **)malloc(n * sizeof(float *));
    for (int j = 0; j < n; j++) {
      (*cube_ref)[i][j] = (float *)malloc(n * sizeof(float));
    }
  }
}

void gen_cube(float ***cube, int n) {
  for (int i = 0; i < n; i++) {
    for (int j = 0; j < n; j++) {
      for (int k = 0; k < n; k++) {
        cube[i][j][k] = (float)((i + j + k) % 10) * (float)1.1;
      }
    }
  }
}

void cpu_calculation(float ***input, float ***output, int n) {
  for (int i = 0; i < n; i++) {
    for (int j = 0; j < n; j++) {
      for (int k = 0; k < n; k++) {
        float elem1 = i > 0 ? input[i - 1][j][k] : 0;
        float elem2 = i < n - 1 ? input[i + 1][j][k] : 0;
        float elem3 = j > 0 ? input[i][j - 1][k] : 0;
        float elem4 = j < n - 1 ? input[i][j + 1][k] : 0;
        float elem5 = k > 0 ? input[i][j][k - 1] : 0;
        float elem6 = k < n - 1 ? input[i][j][k + 1] : 0;

        output[i][j][k] =
            (float)0.8 * (elem1 + elem2 + elem3 + elem4 + elem5 + elem6);
      }
    }
  }
}

float sum(float ***output, int n) {
  float sum = 0;

  for (int i = 0; i < n; i++) {
    for (int j = 0; j < n; j++) {
      for (int k = 0; k < n; k++) {
        sum += output[i][j][k] * (((i + j + k) % 10) ? 1 : -1);
      }
    }
  }

  return sum;
}

int main(int argc, char *argv[]) {
  std::string n_str;
  int n;

  if ((argc != 2)) {
    std::cerr << "Error: wrong number of argument, specify one argument for "
                 "the dimension of the cube.\n";
    return -1;
  } else {
    n_str = argv[1];

    try {
      n = std::stoi(n_str);
    } catch (std::exception &e) {
      std::cerr << "Error, failed to convert n to integer, error "
                   "message:"
                << e.what() << '\n';
      return -1;
    }
  }

  // printf("specified n:%d\n", n);

  float ***input, ***output;
  cpu_malloc_cube(&input, n);
  cpu_malloc_cube(&output, n);
  gen_cube(input, n);
  cpu_calculation(input, output, n);

  float cpu_cal_sum = sum(output, n);
  printf("cpu result sum:%lf\n", cpu_cal_sum);

  return 0;
}