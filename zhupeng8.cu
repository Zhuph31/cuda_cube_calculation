/*
* ECE1782 - W2024 - Lab 2 - Sample Code
* Sample Test Cases (sum)

n, result 
100,18295201.010496
200,147100808.124588
300,497296827.464880
400,1179763265.153962
500,2305380127.308517
600,3985027420.060339
700,6329585154.758305
800,9449933335.045414
*/


#include <hip/hip_runtime.h>
#include <algorithm>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

/*You can use the following for any CUDA function that returns cudaError_t
 * type*/
#define gpuErrchk(ans)                                                         \
  { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line,
                      bool abort = true) {
  if (code == hipSuccess)
    return;

  fprintf(stderr, "Error: %s %s %d\n", hipGetErrorString(code), file, line);
  if (abort)
    exit(code);
}

/*Use the following to get a timestamp*/
double getTimeStamp() {
  struct timeval tv;
  gettimeofday(&tv, NULL);
  return (double)tv.tv_usec / 1000000 + tv.tv_sec;
}

__global__ void jacobiKernel(float *a, float *b, int n, int last_row,
                             int offset) {
  int i = (blockIdx.x * blockDim.x + threadIdx.x) / (n * n) + offset;
  int j = ((blockIdx.x * blockDim.x + threadIdx.x) / n) % n;
  int k = (blockIdx.x * blockDim.x + threadIdx.x) % n;

  if (i >= last_row || j >= n || k >= n)
    return;

  // float newVal = 0.0f;
  // if(i > 0) newVal += b[(i-1)*n*n + j*n + k];
  // if(i < n-1) newVal += b[(i+1)*n*n + j*n + k];
  // if(j > 0) newVal += b[i*n*n + (j-1)*n + k];
  // if(j < n-1) newVal += b[i*n*n + (j+1)*n + k];
  // if(k > 0) newVal += b[i*n*n + j*n + (k-1)];
  // if(k < n-1) newVal += b[i*n*n + j*n + (k+1)];

  // a[i*n*n + j*n + k] = 0.8f * newVal;
  a[i * n * n + j * n + k] =
      0.8f * (((i > 0) ? b[(i - 1) * n * n + j * n + k] : 0) +
              ((i < n - 1) ? b[(i + 1) * n * n + j * n + k] : 0) +
              ((j > 0) ? b[i * n * n + (j - 1) * n + k] : 0) +
              ((j < n - 1) ? b[i * n * n + (j + 1) * n + k] : 0) +
              ((k > 0) ? b[i * n * n + j * n + (k - 1)] : 0) +
              ((k < n - 1) ? b[i * n * n + j * n + (k + 1)] : 0));
}

int main(int argc, char *argv[]) {
  if (argc != 2) {
    printf("Error: wrong number of args\n");
    exit(1);
  }

  int n = atoi(argv[1]);
  size_t number_of_elements = ((size_t)n) * n * n;
  size_t bytes = number_of_elements * sizeof(float);
  int num_streams = std::max(std::min(n / 10, 50), 2) + 1;
  int rows_per_stream = ceil(n / (num_streams - 1.0f));
  int last_stream_rows = n - (num_streams - 2) * rows_per_stream;
  // printf("Number of streams: %d\n", num_streams);
  dim3 threadsPerBlock(1024);
  dim3 blocksPerGrid((rows_per_stream * n * n + 1023) / 1024);

  gpuErrchk(hipDeviceReset());

  float *h_a, *h_b;
  hipHostMalloc(&h_a, bytes, hipHostMallocDefault);
  hipHostMalloc(&h_b, bytes, hipHostMallocDefault);

  // Initialize b_host
  for (int i = 0; i < n; i++) {
    for (int j = 0; j < n; j++) {
      for (int k = 0; k < n; k++) {
        h_b[i * n * n + j * n + k] = (float)((i + j + k) % 10) * 1.1f;
      }
    }
  }

  hipStream_t streams[num_streams];
  for (int i = 1; i < num_streams; i++) {
    hipStreamCreate(&streams[i]);
  }

  hipEvent_t events[num_streams];
  for (int i = 1; i < num_streams; i++) {
    hipEventCreate(&events[i]);
  }

  //================= Timing Begins ========================
  double start_time = getTimeStamp();

  /*Device allocations are included in timing*/
  float *d_a, *d_b;
  hipMalloc(&d_a, bytes);
  hipMalloc(&d_b, bytes);

  hipMemcpyAsync(d_b, h_b, (1 + rows_per_stream) * n * n * sizeof(float),
                  hipMemcpyHostToDevice, streams[1]);
  hipEventRecord(events[1], streams[1]);
  jacobiKernel<<<blocksPerGrid, threadsPerBlock, 0, streams[1]>>>(
      d_a, d_b, n, rows_per_stream, 0);
  hipMemcpyAsync(h_a, d_a, rows_per_stream * n * n * sizeof(float),
                  hipMemcpyDeviceToHost, streams[1]);
  for (int i = 2; i < (num_streams - 1); i++) {
    int offset = (i - 1) * rows_per_stream;
    hipMemcpyAsync(d_b + (offset + 1) * n * n, h_b + (offset + 1) * n * n,
                    rows_per_stream * n * n * sizeof(float),
                    hipMemcpyHostToDevice, streams[i]);
    hipEventRecord(events[i], streams[i]);
    hipStreamWaitEvent(streams[i], events[i - 1], 0);
    jacobiKernel<<<blocksPerGrid, threadsPerBlock, 0, streams[i]>>>(
        d_a, d_b, n, i * rows_per_stream, offset);
    hipMemcpyAsync(h_a + offset * n * n, d_a + offset * n * n,
                    rows_per_stream * n * n * sizeof(float),
                    hipMemcpyDeviceToHost, streams[i]);
  }
  if (last_stream_rows > 0) {
    int offset = (num_streams - 2) * rows_per_stream;
    hipMemcpyAsync(d_b + (offset + 1) * n * n, h_b + (offset + 1) * n * n,
                    (last_stream_rows - 1) * n * n * sizeof(float),
                    hipMemcpyHostToDevice, streams[num_streams - 1]);
    hipStreamWaitEvent(streams[num_streams - 1], events[num_streams - 2], 0);
    jacobiKernel<<<blocksPerGrid, threadsPerBlock, 0,
                   streams[num_streams - 1]>>>(d_a, d_b, n, n, offset);
    hipMemcpyAsync(h_a + offset * n * n, d_a + offset * n * n,
                    last_stream_rows * n * n * sizeof(float),
                    hipMemcpyDeviceToHost, streams[num_streams - 1]);
  }
  hipDeviceSynchronize();
  double end_time = getTimeStamp();
  //================= Timing Ends ========================    
  int total_time_ms = (int)ceil((end_time - start_time) * 1000);
  double sum = 0.0;
  // compute a at host side and check the result
  // float *a_host = (float *)malloc(bytes);
  // for(int i = 0; i < n; i++) {
  //     for(int j = 0; j < n; j++) {
  //         for(int k = 0; k < n; k++) {
  //             if (i > 0) a_host[i*n*n + j*n + k] += h_b[(i-1)*n*n + j*n + k];
  //             if (i < n-1) a_host[i*n*n + j*n + k] += h_b[(i+1)*n*n + j*n +
  //             k]; if (j > 0) a_host[i*n*n + j*n + k] += h_b[i*n*n + (j-1)*n +
  //             k]; if (j < n-1) a_host[i*n*n + j*n + k] += h_b[i*n*n + (j+1)*n
  //             + k]; if (k > 0) a_host[i*n*n + j*n + k] += h_b[i*n*n + j*n +
  //             (k-1)]; if (k < n-1) a_host[i*n*n + j*n + k] += h_b[i*n*n + j*n
  //             + (k+1)]; a_host[i*n*n + j*n + k] *= 0.8f; if (a_host[i*n*n +
  //             j*n + k] != h_a[i*n*n + j*n + k]) {
  //                 printf("Mismatch at %d %d %d: %f %f\n", i, j, k,
  //                 a_host[i*n*n + j*n + k], h_a[i*n*n + j*n + k]);
  //             }
  //         }
  //     }
  // }
  for (int i = 0; i < n; i++) {
    for (int j = 0; j < n; j++) {
      for (int k = 0; k < n; k++) {
        sum += h_a[i * n * n + j * n + k] * (((i + j + k) % 10) ? 1 : -1);
      }
    }
  }

  printf("%lf %d\n", sum, total_time_ms);

  for (int i = 1; i < num_streams; i++) {
    hipStreamDestroy(streams[i]);
  }
  for (int i = 1; i < num_streams; i++) {
    hipEventDestroy(events[i]);
  }
  hipFree(d_a);
  hipFree(d_b);
  hipHostFree(h_a);
  hipHostFree(h_b);
  gpuErrchk(hipDeviceReset());
  return 0;
}
